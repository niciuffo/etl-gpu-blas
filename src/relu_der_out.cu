#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/relu_der_out.hpp"

template <typename T>
__global__ void relu_der_out_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = x[incx * index] > T(0) ? alpha : T(0);
     }
}

template <typename T>
__global__ void relu_der_out_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = x[incx * index] > T(0) ? T(1) : T(0);
    }
}

template <typename T>
__global__ void relu_der_out_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = T(0);
    }
}

template <typename T>
void relu_der_out_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, relu_der_out_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    relu_der_out_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void relu_der_out_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, relu_der_out_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    relu_der_out_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void relu_der_out_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, relu_der_out_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    relu_der_out_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

    hipDeviceSynchronize();
}

void egblas_srelu_der_out(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        relu_der_out_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        relu_der_out_kernel0_run(n, y, incy);
    } else {
        relu_der_out_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_drelu_der_out(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        relu_der_out_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        relu_der_out_kernel0_run(n, y, incy);
    } else {
        relu_der_out_kernel_run(n, alpha, x, incx, y, incy);
    }
}
