#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

#include "egblas/assert.hpp"
#include "egblas/utils.hpp"
#include "egblas/sum.hpp"
#include "egblas/cuda_check.hpp"

#include "sum_reduce.hpp"

template <class T, size_t blockSize>
__global__ void sum_kernel(size_t n, const T* input, size_t incx, T* output) {
    extern __shared__ __align__(sizeof(T)) volatile unsigned char shared_data_raw[];

    volatile T* shared_data = reinterpret_cast<volatile T*>(shared_data_raw);

    size_t tid      = threadIdx.x;
    size_t i        = blockIdx.x * (2 * blockDim.x) + threadIdx.x;
    size_t gridSize = blockSize * 2 * gridDim.x;

    // Perform first level of durection,
    // reading from global memory and writing to shared memory

    T mySum = 0;

    while (i < n) {
        mySum += input[i * incx];

        if (i + blockSize < n) {
            mySum += input[(i + blockSize) * incx];
        }

        i += gridSize;
    }

    shared_data[tid] = mySum;

    __syncthreads();

    sum_reduce_impl<T, blockSize>(output, shared_data, mySum);
}

template <typename T>
void invoke_sum_kernel(size_t n, const T* input, size_t incx, T* output, size_t numThreads, size_t numBlocks) {
    int sharedSize = (numThreads <= 32) ? 64 * sizeof(T) : numThreads * sizeof(T);

    switch (numThreads) {
        case 512:
            sum_kernel<T, 512><<<numBlocks, numThreads, sharedSize>>>(n, input, incx, output);
            break;

        case 256:
            sum_kernel<T, 256><<<numBlocks, numThreads, sharedSize>>>(n, input, incx, output);
            break;

        case 128:
            sum_kernel<T, 128><<<numBlocks, numThreads, sharedSize>>>(n, input, incx, output);
            break;

        case 64:
            sum_kernel<T,  64><<<numBlocks, numThreads, sharedSize>>>(n, input, incx, output);
            break;

        case 32:
            sum_kernel<T,  32><<<numBlocks, numThreads, sharedSize>>>(n, input, incx, output);
            break;

        case 16:
            sum_kernel<T,  16><<<numBlocks, numThreads, sharedSize>>>(n, input, incx, output);
            break;

        case 8:
            sum_kernel<T,   8><<<numBlocks, numThreads, sharedSize>>>(n, input, incx, output);
            break;

        case 4:
            sum_kernel<T,   4><<<numBlocks, numThreads, sharedSize>>>(n, input, incx, output);
            break;

        case 2:
            sum_kernel<T,   2><<<numBlocks, numThreads, sharedSize>>>(n, input, incx, output);
            break;

        case 1:
            sum_kernel<T,   1><<<numBlocks, numThreads, sharedSize>>>(n, input, incx, output);
            break;
    }
}

template <typename T>
T sum_kernel_run(size_t n, const T* input, size_t incx) {
    T result = 0;

    const size_t cpu_threshold = 1024;

    if (n <= cpu_threshold && incx == 1) {
        if (n > 1) {
            T* host_data = new T[n];

            cuda_check(hipMemcpy(host_data, input, n * sizeof(T), hipMemcpyDeviceToHost));

            for (size_t i = 0; i < n; i++) {
                result += host_data[i];
            }

            delete[] host_data;
        } else {
            cuda_check(hipMemcpy(&result, input, 1 * sizeof(T), hipMemcpyDeviceToHost));
        }

        return result;
    }

    const size_t maxThreads    = 256;
    const size_t maxBlocks     = 64;

    // Compute the launch configuration of the kernel
    size_t numThreads = n < maxThreads * 2 ? nextPow2((n + 1) / 2) : maxThreads;
    size_t numBlocks  = std::min((n + numThreads * 2 - 1) / (numThreads * 2), maxBlocks);

    // Allocate memory on the device

    T* y_gpu_1;
    T* y_gpu_2;
    cuda_check(hipMalloc((void**)&y_gpu_1, numBlocks * sizeof(T)));
    cuda_check(hipMalloc((void**)&y_gpu_2, numBlocks * sizeof(T)));

    hipMemset(y_gpu_1, 0, numBlocks * sizeof(T));
    hipMemset(y_gpu_2, 0, numBlocks * sizeof(T));

    // Run the first reduction on GPU

    invoke_sum_kernel<T>(n, input, incx, y_gpu_2, numThreads, numBlocks);

    size_t s = numBlocks;

    // Run the following reductions on GPU

    while(s > cpu_threshold){
        // Compute again the configuration of the reduction kernel
        numThreads = n < maxThreads * 2 ? nextPow2((n + 1) / 2) : maxThreads;
        numBlocks  = std::min((n + numThreads * 2 - 1) / (numThreads * 2), maxBlocks);

        cuda_check(hipMemcpy(y_gpu_1, y_gpu_2, s * sizeof(T), hipMemcpyDeviceToDevice));

        invoke_sum_kernel<T>(s, y_gpu_1, 1, y_gpu_2, numThreads, numBlocks);

        s = (s + numThreads * 2 - 1) / (numThreads * 2);
    }

    if(s > 1){
        T* host_data = new T[s];

        cuda_check(hipMemcpy(host_data, y_gpu_2, s * sizeof(T), hipMemcpyDeviceToHost));

        for (size_t i = 0; i < s; i++) {
            result += host_data[i];
        }

        delete[] host_data;
    } else {
        cuda_check(hipMemcpy(&result, y_gpu_2, 1 * sizeof(T), hipMemcpyDeviceToHost));
    }

    cuda_check(hipFree(y_gpu_1));
    cuda_check(hipFree(y_gpu_2));

    return result;
}

float egblas_ssum(float* x, size_t n, size_t s) {
    egblas_assert(s == 1, "Stride is not yet supported for egblas_ssum");
    egblas_unused(s);

    return sum_kernel_run(n, x, s);
}

double egblas_dsum(double* x, size_t n, size_t s) {
    egblas_assert(s == 1, "Stride is not yet supported for egblas_dsum");
    egblas_unused(s);

    return sum_kernel_run(n, x, s);
}

// Complex sums are done with thrust

struct single_complex_add {
    __device__ hipComplex operator()(hipComplex x, hipComplex y) {
        return hipCaddf(x, y);
    }
};

struct double_complex_add {
    __device__ hipDoubleComplex operator()(hipDoubleComplex x, hipDoubleComplex y) {
        return hipCadd(x, y);
    }
};

hipComplex egblas_csum(hipComplex* x, size_t n, size_t s) {
    egblas_assert(s == 1, "Stride is not yet supported for egblas_csum");
    egblas_unused(s);

    return thrust::reduce(thrust::device, x, x + n, make_hipComplex(0, 0), single_complex_add());
}

hipDoubleComplex egblas_zsum(hipDoubleComplex* x, size_t n, size_t s) {
    egblas_assert(s == 1, "Stride is not yet supported for egblas_zsum");
    egblas_unused(s);

    return thrust::reduce(thrust::device, x, x + n, make_hipDoubleComplex(0, 0), double_complex_add());
}
