#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/scalar_set.hpp"

template <typename T>
__global__ void scalar_set_kernel(T* x, size_t n, size_t s, const T beta) {
    auto index  = 1 * (threadIdx.x + blockIdx.x * blockDim.x);
    auto stride = 1 * (blockDim.x * gridDim.x);

    for (; index < n; index += stride) {
        x[s * index] = beta;
    }
}

template <typename T>
void scalar_set_kernel_run(T* x, size_t n, size_t s, T beta) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, scalar_set_kernel<T>, 0, 0);

    int gridSize = ((n / s) + blockSize - 1) / blockSize;

    scalar_set_kernel<T><<<gridSize, blockSize>>>(x, n, s, beta);

    hipDeviceSynchronize();
}

void egblas_scalar_sset(float* x, size_t n, size_t s, float beta) {
    scalar_set_kernel_run(x, n, s, beta);
}

void egblas_scalar_dset(double* x, size_t n, size_t s, double beta) {
    scalar_set_kernel_run(x, n, s, beta);
}
