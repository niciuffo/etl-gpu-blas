#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/sinh.hpp"

template <typename T>
__global__ void sinh_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * sinh(x[incx * index]);
    }
}

template <>
__global__ void sinh_kernel(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        auto res = make_hipComplex(sinh(c.x) * cos(c.y), cosh(c.x) * sin(c.y));

        y[incx * index] = hipCmulf(alpha, res);
    }
}

template <>
__global__ void sinh_kernel(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        auto res = make_hipDoubleComplex(sinh(c.x) * cos(c.y), cosh(c.x) * sin(c.y));

        y[incx * index] = hipCmul(alpha, res);
    }
}

template <typename T>
__global__ void sinh_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = sinh(x[incx * index]);
    }
}

template <>
__global__ void sinh_kernel1(size_t n, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        auto res = make_hipComplex(sinh(c.x) * cos(c.y), cosh(c.x) * sin(c.y));

        y[incx * index] = res;
    }
}

template <>
__global__ void sinh_kernel1(size_t n, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        auto res = make_hipDoubleComplex(sinh(c.x) * cos(c.y), cosh(c.x) * sin(c.y));

        y[incx * index] = res;
    }
}

template <typename T>
__global__ void sinh_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = T(0);
    }
}

template <>
__global__ void sinh_kernel0(size_t n, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipComplex(0, 0);
    }
}

template <>
__global__ void sinh_kernel0(size_t n, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipDoubleComplex(0, 0);
    }
}

template <typename T>
void sinh_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, sinh_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    sinh_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void sinh_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, sinh_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    sinh_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void sinh_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, sinh_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    sinh_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

    hipDeviceSynchronize();
}

void egblas_ssinh(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        sinh_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        sinh_kernel0_run(n, y, incy);
    } else {
        sinh_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dsinh(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        sinh_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        sinh_kernel0_run(n, y, incy);
    } else {
        sinh_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_csinh(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        sinh_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        sinh_kernel0_run(n, y, incy);
    } else {
        sinh_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zsinh(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        sinh_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        sinh_kernel0_run(n, y, incy);
    } else {
        sinh_kernel_run(n, alpha, x, incx, y, incy);
    }
}
