#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/tan.hpp"

template <typename T>
__global__ void tan_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * tan(x[incx * index]);
    }
}

template <>
__global__ void tan_kernel(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        auto res_sin = make_hipComplex(sin(c.x) * cosh(c.y), cos(c.x) * sinh(c.y));
        auto res_cos = make_hipComplex(cos(c.x) * cosh(c.y), -sin(c.x) * sinh(c.y));
        auto res = hipCdivf(res_sin, res_cos);

        y[incx * index] = hipCmulf(alpha, res);
    }
}

template <>
__global__ void tan_kernel(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        auto res_sin = make_hipDoubleComplex(sin(c.x) * cosh(c.y), cos(c.x) * sinh(c.y));
        auto res_cos = make_hipDoubleComplex(cos(c.x) * cosh(c.y), -sin(c.x) * sinh(c.y));
        auto res = hipCdiv(res_sin, res_cos);

        y[incx * index] = hipCmul(alpha, res);
    }
}

template <typename T>
__global__ void tan_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = tan(x[incx * index]);
    }
}

template <>
__global__ void tan_kernel1(size_t n, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        auto res_sin = make_hipComplex(sin(c.x) * cosh(c.y), cos(c.x) * sinh(c.y));
        auto res_cos = make_hipComplex(cos(c.x) * cosh(c.y), -sin(c.x) * sinh(c.y));
        auto res = hipCdivf(res_sin, res_cos);

        y[incx * index] = res;
    }
}

template <>
__global__ void tan_kernel1(size_t n, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        auto res_sin = make_hipDoubleComplex(sin(c.x) * cosh(c.y), cos(c.x) * sinh(c.y));
        auto res_cos = make_hipDoubleComplex(cos(c.x) * cosh(c.y), -sin(c.x) * sinh(c.y));
        auto res = hipCdiv(res_sin, res_cos);

        y[incx * index] = res;
    }
}

template <typename T>
__global__ void tan_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = T(0);
    }
}

template <>
__global__ void tan_kernel0(size_t n, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipComplex(0, 0);
    }
}

template <>
__global__ void tan_kernel0(size_t n, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipDoubleComplex(0, 0);
    }
}

template <typename T>
void tan_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, tan_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    tan_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void tan_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, tan_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    tan_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void tan_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, tan_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    tan_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_stan(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        tan_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        tan_kernel0_run(n, y, incy);
    } else {
        tan_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dtan(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        tan_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        tan_kernel0_run(n, y, incy);
    } else {
        tan_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_ctan(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        tan_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        tan_kernel0_run(n, y, incy);
    } else {
        tan_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_ztan(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        tan_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        tan_kernel0_run(n, y, incy);
    } else {
        tan_kernel_run(n, alpha, x, incx, y, incy);
    }
}
