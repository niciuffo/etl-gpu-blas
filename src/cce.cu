#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include <iostream>

#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

#include "egblas/cce.hpp"
#include "egblas/cuda_check.hpp"

template <typename T>
__global__ void cce_loss_kernel(size_t n, const T* output, size_t incx, const T* labels, size_t incy, T* y) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = logf(output[incx * index]) * labels[incx * index];
    }
}

template <typename T>
__global__ void cce_error_kernel(size_t n, size_t m, const T* output, const T* labels, T* y) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto i = index;

        int max_l = 0;
        int max_o = 0;

        // Compute the max for argmax

        for (size_t j = 1; j < m; ++j) {
            if (labels[i * m + j] > labels[i * m + max_l]) {
                max_l = j;
            }

            if (output[i * m + j] > output[i * m + max_o]) {
                max_o = j;
            }
        }

        // Compute the final value

        y[i] = fmin(abs(max_l - max_o), T(1.0));
    }
}

template <typename T>
void cce_loss_kernel_run(size_t n, const T* output, size_t incx, const T* labels, size_t incy, T* y) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cce_loss_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    cce_loss_kernel<T><<<gridSize, blockSize>>>(n, output, incx, labels, incy, y);

    hipDeviceSynchronize();
}

template <typename T>
void cce_error_kernel_run(size_t n, size_t m, const T* output, const T* labels, T* y) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cce_loss_kernel<T>, 0, 0);

    int gridSize = (n + blockSize - 1) / blockSize;

    cce_error_kernel<T><<<gridSize, blockSize>>>(n, m, output, labels, y);

    hipDeviceSynchronize();
}

float egblas_cce_sloss(size_t n, float alpha, const float* output, size_t incx, const float* labels, size_t incy) {
    float* temp;
    cuda_check(hipMalloc((void**)&temp, n * sizeof(float)));

    cce_loss_kernel_run(n, output, incx, labels, incy, temp);

    float loss = thrust::reduce(thrust::device, temp, temp + n);

    cuda_check(hipFree(temp));

    return alpha * loss;
}

double egblas_cce_dloss(size_t n, double alpha, const double* output, size_t incx, const double* labels, size_t incy) {
    double* temp;
    cuda_check(hipMalloc((void**)&temp, n * sizeof(double)));

    cce_loss_kernel_run(n, output, incx, labels, incy, temp);

    double loss = thrust::reduce(thrust::device, temp, temp + n);

    cuda_check(hipFree(temp));

    return alpha * loss;
}

float egblas_cce_serror(size_t n, size_t m, float alpha, const float* output, const float* labels) {
    float* temp;
    cuda_check(hipMalloc((void**)&temp, n * sizeof(float)));

    cce_error_kernel_run(n, m, output, labels, temp);

    float loss = thrust::reduce(thrust::device, temp, temp + n);

    cuda_check(hipFree(temp));

    return alpha * loss * 1.0f;
}

double egblas_cce_derror(size_t n, size_t m, double alpha, const double* output, const double* labels) {
    double* temp;
    cuda_check(hipMalloc((void**)&temp, n * sizeof(double)));

    cce_error_kernel_run(n, m, output, labels, temp);

    double loss = thrust::reduce(thrust::device, temp, temp + n);

    cuda_check(hipFree(temp));

    return alpha * loss;
}
