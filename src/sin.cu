#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/sin.hpp"

template <typename T>
__global__ void sin_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * sin(x[incx * index]);
    }
}

template <>
__global__ void sin_kernel(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        auto res = make_hipComplex(sin(c.x) * cosh(c.y), cos(c.x) * sinh(c.y));

        y[incx * index] = hipCmulf(alpha, res);
    }
}

template <>
__global__ void sin_kernel(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        auto res = make_hipDoubleComplex(sin(c.x) * cosh(c.y), cos(c.x) * sinh(c.y));

        y[incx * index] = hipCmul(alpha, res);
    }
}

template <typename T>
__global__ void sin_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = sin(x[incx * index]);
    }
}

template <>
__global__ void sin_kernel1(size_t n, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        auto res = make_hipComplex(sin(c.x) * cosh(c.y), cos(c.x) * sinh(c.y));

        y[incx * index] = res;
    }
}

template <>
__global__ void sin_kernel1(size_t n, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        auto res = make_hipDoubleComplex(sin(c.x) * cosh(c.y), cos(c.x) * sinh(c.y));

        y[incx * index] = res;
    }
}

template <typename T>
__global__ void sin_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = T(0);
    }
}

template <>
__global__ void sin_kernel0(size_t n, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipComplex(0, 0);
    }
}

template <>
__global__ void sin_kernel0(size_t n, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipDoubleComplex(0, 0);
    }
}

template <typename T>
void sin_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, sin_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    sin_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void sin_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, sin_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    sin_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void sin_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, sin_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    sin_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

    hipDeviceSynchronize();
}

void egblas_ssin(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        sin_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        sin_kernel0_run(n, y, incy);
    } else {
        sin_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dsin(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        sin_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        sin_kernel0_run(n, y, incy);
    } else {
        sin_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_csin(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        sin_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        sin_kernel0_run(n, y, incy);
    } else {
        sin_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zsin(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        sin_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        sin_kernel0_run(n, y, incy);
    } else {
        sin_kernel_run(n, alpha, x, incx, y, incy);
    }
}
