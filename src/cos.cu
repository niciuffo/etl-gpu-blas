#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/cos.hpp"

template <typename T>
__global__ void cos_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * cos(x[incx * index]);
    }
}

template <>
__global__ void cos_kernel(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        auto res = make_hipComplex(cos(c.x) * cosh(c.y), -sin(c.x) * sinh(c.y));

        y[incx * index] = hipCmulf(alpha, res);
    }
}

template <>
__global__ void cos_kernel(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        auto res = make_hipDoubleComplex(cos(c.x) * cosh(c.y), -sin(c.x) * sinh(c.y));

        y[incx * index] = hipCmul(alpha, res);
    }
}

template <typename T>
__global__ void cos_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = cos(x[incx * index]);
    }
}

template <>
__global__ void cos_kernel1(size_t n, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        auto res = make_hipComplex(cos(c.x) * cosh(c.y), -sin(c.x) * sinh(c.y));

        y[incx * index] = res;
    }
}

template <>
__global__ void cos_kernel1(size_t n, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        auto res = make_hipDoubleComplex(cos(c.x) * cosh(c.y), -sin(c.x) * sinh(c.y));

        y[incx * index] = res;
    }
}

template <typename T>
__global__ void cos_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = T(0);
    }
}

template <>
__global__ void cos_kernel0(size_t n, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipComplex(0, 0);
    }
}

template <>
__global__ void cos_kernel0(size_t n, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipDoubleComplex(0, 0);
    }
}

template <typename T>
void cos_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cos_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    cos_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void cos_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cos_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    cos_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void cos_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cos_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    cos_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

    hipDeviceSynchronize();
}

void egblas_scos(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        cos_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        cos_kernel0_run(n, y, incy);
    } else {
        cos_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dcos(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        cos_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        cos_kernel0_run(n, y, incy);
    } else {
        cos_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_ccos(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        cos_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        cos_kernel0_run(n, y, incy);
    } else {
        cos_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zcos(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        cos_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        cos_kernel0_run(n, y, incy);
    } else {
        cos_kernel_run(n, alpha, x, incx, y, incy);
    }
}
