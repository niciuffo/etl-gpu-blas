#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/log.hpp"

template <typename T>
__global__ void log_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * logf(x[incx * index]);
    }
}

template <typename T>
__global__ void log_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = logf(x[incx * index]);
    }
}

template <typename T>
__global__ void log_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = T(0);
    }
}

template <typename T>
void log_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, log_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    log_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void log_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, log_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    log_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void log_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, log_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    log_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

    hipDeviceSynchronize();
}

void egblas_slog(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        log_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        log_kernel0_run(n, y, incy);
    } else {
        log_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dlog(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        log_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        log_kernel0_run(n, y, incy);
    } else {
        log_kernel_run(n, alpha, x, incx, y, incy);
    }
}
