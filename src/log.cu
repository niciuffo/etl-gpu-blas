#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/log.hpp"

template <typename T>
__global__ void log_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * log(x[incx * index]);
    }
}

template <>
__global__ void log_kernel(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        float c_abs = hypot(c.x, c.y);
        float c_arg = atan2(c.y, c.x);

        y[incx * index] = hipCmulf(alpha, make_hipComplex(log(c_abs), c_arg));
    }
}

template <>
__global__ void log_kernel(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        double c_abs = hypot(c.x, c.y);
        double c_arg = atan2(c.y, c.x);

        y[incx * index] = hipCmul(alpha, make_hipDoubleComplex(log(c_abs), c_arg));
    }
}

template <typename T>
__global__ void log_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = log(x[incx * index]);
    }
}

template <>
__global__ void log_kernel1(size_t n, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        float c_abs = hypot(c.x, c.y);
        float c_arg = atan2(c.y, c.x);

        y[incx * index] = make_hipComplex(log(c_abs), c_arg);
    }
}

template <>
__global__ void log_kernel1(size_t n, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        double c_abs = hypot(c.x, c.y);
        double c_arg = atan2(c.y, c.x);

        y[incx * index] = make_hipDoubleComplex(log(c_abs), c_arg);
    }
}

template <typename T>
__global__ void log_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = T(0);
    }
}

template <>
__global__ void log_kernel0(size_t n, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipComplex(0, 0);
    }
}

template <>
__global__ void log_kernel0(size_t n, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipDoubleComplex(0, 0);
    }
}

template <typename T>
void log_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, log_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    log_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void log_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, log_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    log_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void log_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, log_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    log_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

    hipDeviceSynchronize();
}

void egblas_slog(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        log_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        log_kernel0_run(n, y, incy);
    } else {
        log_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dlog(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        log_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        log_kernel0_run(n, y, incy);
    } else {
        log_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_clog(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        log_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        log_kernel0_run(n, y, incy);
    } else {
        log_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zlog(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        log_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        log_kernel0_run(n, y, incy);
    } else {
        log_kernel_run(n, alpha, x, incx, y, incy);
    }
}
