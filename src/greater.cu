#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/greater.hpp"

#include "complex.hpp"

__device__ bool operator>(hipComplex lhs, hipComplex rhs){
    return lhs.x > rhs.x || (lhs.x == rhs.x && lhs.y > rhs.y);
}

__device__ bool operator>(hipDoubleComplex lhs, hipDoubleComplex rhs){
    return lhs.x > rhs.x || (lhs.x == rhs.x && lhs.y > rhs.y);
}

template <typename T>
__global__ void greater_kernel(size_t n, const T* a, size_t inca, const T* b, size_t incb, bool* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = a[inca * index] > b[incb * index];
    }
}

template <typename T>
void greater_kernel_run(size_t n, const T* a, size_t inca, const T* b, size_t incb, bool* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, greater_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    greater_kernel<T><<<gridSize, blockSize>>>(n, a, inca, b, incb, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_sgreater(size_t n, const float* a, size_t inca, const float* b, size_t incb, bool* y, size_t incy) {
    greater_kernel_run(n, a, inca, b, incb, y, incy);
}

void egblas_dgreater(size_t n, const double* a, size_t inca, const double* b, size_t incb, bool* y, size_t incy) {
    greater_kernel_run(n, a, inca, b, incb, y, incy);
}

void egblas_cgreater(size_t n, const hipComplex* a, size_t inca, const hipComplex* b, size_t incb, bool* y, size_t incy) {
    greater_kernel_run(n, a, inca, b, incb, y, incy);
}

void egblas_zgreater(size_t n, const hipDoubleComplex* a, size_t inca, const hipDoubleComplex* b, size_t incb, bool* y, size_t incy) {
    greater_kernel_run(n, a, inca, b, incb, y, incy);
}
