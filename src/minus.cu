#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/minus.hpp"

template <typename T>
__global__ void minus_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * -(x[incx * index]);
    }
}

template <>
__global__ void minus_kernel(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        auto res = make_hipComplex(-c.x, -c.y);

        y[incx * index] = hipCmulf(alpha, res);
    }
}

template <>
__global__ void minus_kernel(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        auto res = make_hipDoubleComplex(-c.x, -c.y);

        y[incx * index] = hipCmul(alpha, res);
    }
}

template <typename T>
__global__ void minus_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = -(x[incx * index]);
    }
}

template <>
__global__ void minus_kernel1(size_t n, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        auto res = make_hipComplex(-c.x, -c.y);

        y[incx * index] = res;
    }
}

template <>
__global__ void minus_kernel1(size_t n, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        auto res = make_hipDoubleComplex(-c.x, -c.y);

        y[incx * index] = res;
    }
}

template <typename T>
__global__ void minus_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = T(0);
    }
}

template <>
__global__ void minus_kernel0(size_t n, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipComplex(0, 0);
    }
}

template <>
__global__ void minus_kernel0(size_t n, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipDoubleComplex(0, 0);
    }
}

template <typename T>
void minus_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, minus_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    minus_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void minus_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, minus_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    minus_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void minus_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, minus_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    minus_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_sminus(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        minus_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        minus_kernel0_run(n, y, incy);
    } else {
        minus_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dminus(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        minus_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        minus_kernel0_run(n, y, incy);
    } else {
        minus_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_cminus(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        minus_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        minus_kernel0_run(n, y, incy);
    } else {
        minus_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zminus(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        minus_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        minus_kernel0_run(n, y, incy);
    } else {
        minus_kernel_run(n, alpha, x, incx, y, incy);
    }
}
