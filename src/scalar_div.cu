#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/scalar_add.hpp"

template <typename T>
__global__ void scalar_div_kernel(const T beta, T* x, size_t n, size_t s) {
    auto index  = 1 * (threadIdx.x + blockIdx.x * blockDim.x);
    auto stride = 1 * (blockDim.x * gridDim.x);

    for (; index < n; index += stride) {
        x[s * index] = beta / x[s * index];
    }
}

template <>
__global__ void scalar_div_kernel(const hipComplex beta, hipComplex* x, size_t n, size_t s) {
    auto index  = 1 * (threadIdx.x + blockIdx.x * blockDim.x);
    auto stride = 1 * (blockDim.x * gridDim.x);

    for (; index < n; index += stride) {
        x[s * index] = hipCdivf(beta, x[s * index]);
    }
}

template <>
__global__ void scalar_div_kernel(const hipDoubleComplex beta, hipDoubleComplex* x, size_t n, size_t s) {
    auto index  = 1 * (threadIdx.x + blockIdx.x * blockDim.x);
    auto stride = 1 * (blockDim.x * gridDim.x);

    for (; index < n; index += stride) {
        x[s * index] = hipCdiv(beta, x[s * index]);
    }
}

template <typename T>
void scalar_div_kernel_run(T beta, T* x, size_t n, size_t s) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, scalar_div_kernel<T>, 0, 0);

    int gridSize = ((n / s) + blockSize - 1) / blockSize;

    scalar_div_kernel<T><<<gridSize, blockSize>>>(beta, x, n, s);

    hipDeviceSynchronize();
}

void egblas_scalar_sdiv(float beta, float* x, size_t n, size_t s) {
    scalar_div_kernel_run(beta, x, n, s);
}

void egblas_scalar_ddiv(double beta, double* x, size_t n, size_t s) {
    scalar_div_kernel_run(beta, x, n, s);
}

void egblas_scalar_cdiv(hipComplex beta, hipComplex* x, size_t n, size_t s) {
    scalar_div_kernel_run(beta, x, n, s);
}

void egblas_scalar_zdiv(hipDoubleComplex beta, hipDoubleComplex* x, size_t n, size_t s) {
    scalar_div_kernel_run(beta, x, n, s);
}
