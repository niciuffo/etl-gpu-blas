#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/pow.hpp"

template <typename T>
__global__ void pow_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * pow(x[incx * index], y[incy * index]);
    }
}

template <>
__global__ void pow_kernel(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto x_ =  x[incx * index];
        auto y_ =  y[incy * index];

        float c_abs = hypot(x_.x, x_.y);
        float c_arg = atan2(x_.y, x_.x);

        auto logx = make_hipComplex(log(c_abs), c_arg);
        auto ylogx = hipCmulf(y_, logx);

        float e = exp(ylogx.x);
        auto res = make_hipComplex(e * cos(ylogx.y), e * sin(ylogx.y));

        y[incy * index] = hipCmulf(alpha, res);
    }
}

template <>
__global__ void pow_kernel(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto x_ =  x[incx * index];
        auto y_ =  y[incy * index];

        double c_abs = hypot(x_.x, x_.y);
        double c_arg = atan2(x_.y, x_.x);

        auto logx = make_hipDoubleComplex(log(c_abs), c_arg);
        auto ylogx = hipCmul(y_, logx);

        double e = exp(ylogx.x);
        auto res = make_hipDoubleComplex(e * cos(ylogx.y), e * sin(ylogx.y));

        y[incy * index] = hipCmul(alpha, res);
    }
}

template <typename T>
__global__ void pow_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = pow(x[incx * index], y[incy * index]);
    }
}

template <>
__global__ void pow_kernel1(size_t n, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto x_ =  x[incx * index];
        auto y_ =  y[incy * index];

        float c_abs = hypot(x_.x, x_.y);
        float c_arg = atan2(x_.y, x_.x);

        auto logx = make_hipComplex(log(c_abs), c_arg);

        auto ylogx = hipCmulf(y_, logx);

        float e = exp(ylogx.x);
        auto res = make_hipComplex(e * cos(ylogx.y), e * sin(ylogx.y));

        y[incy * index] = res;
    }
}

template <>
__global__ void pow_kernel1(size_t n, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto x_ =  x[incx * index];
        auto y_ =  y[incy * index];

        double c_abs = hypot(x_.x, x_.y);
        double c_arg = atan2(x_.y, x_.x);

        auto logx = make_hipDoubleComplex(log(c_abs), c_arg);
        auto ylogx = hipCmul(y_, logx);

        double e = exp(ylogx.x);
        auto res = make_hipDoubleComplex(e * cos(ylogx.y), e * sin(ylogx.y));

        y[incy * index] = res;
    }
}

template <typename T>
__global__ void pow_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = T(0);
    }
}

template <>
__global__ void pow_kernel0(size_t n, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipComplex(0, 0);
    }
}

template <>
__global__ void pow_kernel0(size_t n, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipDoubleComplex(0, 0);
    }
}

template <typename T>
void pow_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, pow_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    pow_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void pow_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, pow_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    pow_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void pow_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, pow_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    pow_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

    hipDeviceSynchronize();
}

void egblas_spow(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        pow_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        pow_kernel0_run(n, y, incy);
    } else {
        pow_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dpow(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        pow_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        pow_kernel0_run(n, y, incy);
    } else {
        pow_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_cpow(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        pow_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        pow_kernel0_run(n, y, incy);
    } else {
        pow_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zpow(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        pow_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        pow_kernel0_run(n, y, incy);
    } else {
        pow_kernel_run(n, alpha, x, incx, y, incy);
    }
}
