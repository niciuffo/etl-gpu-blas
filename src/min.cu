#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/min.hpp"

template <typename T>
__global__ void min_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * min(x[incx * index], y[incy * index]);
    }
}

template <>
__global__ void min_kernel(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto x_ =  x[incx * index];
        auto y_ =  y[incy * index];

        if (x_.x > y_.x) {
            y[incy * index] = hipCmulf(alpha, y_);
        } else if (y_.x > x_.x) {
            y[incy * index] = hipCmulf(alpha, x_);
        } else {
            if (x_.y > y_.y) {
                y[incy * index] = hipCmulf(alpha, y_);
            } else {
                y[incy * index] = hipCmulf(alpha, x_);
            }
        }
    }
}

template <>
__global__ void min_kernel(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto x_ =  x[incx * index];
        auto y_ =  y[incy * index];

        if (x_.x > y_.x) {
            y[incy * index] = hipCmul(alpha, y_);
        } else if (y_.x > x_.x) {
            y[incy * index] = hipCmul(alpha, x_);
        } else {
            if (x_.y > y_.y) {
                y[incy * index] = hipCmul(alpha, y_);
            } else {
                y[incy * index] = hipCmul(alpha, x_);
            }
        }
    }
}

template <typename T>
__global__ void min_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = min(x[incx * index], y[incy * index]);
    }
}

template <>
__global__ void min_kernel1(size_t n, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto x_ =  x[incx * index];
        auto y_ =  y[incy * index];

        if (x_.x > y_.x) {
            y[incy * index] = y_;
        } else if (y_.x > x_.x) {
            y[incy * index] = x_;
        } else {
            if (x_.y > y_.y) {
                y[incy * index] = y_;
            } else {
                y[incy * index] = x_;
            }
        }
    }
}

template <>
__global__ void min_kernel1(size_t n, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto x_ =  x[incx * index];
        auto y_ =  y[incy * index];

        if (x_.x > y_.x) {
            y[incy * index] = y_;
        } else if (y_.x > x_.x) {
            y[incy * index] = x_;
        } else {
            if (x_.y > y_.y) {
                y[incy * index] = y_;
            } else {
                y[incy * index] = x_;
            }
        }
    }
}

template <typename T>
__global__ void min_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = T(0);
    }
}

template <>
__global__ void min_kernel0(size_t n, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipComplex(0, 0);
    }
}

template <>
__global__ void min_kernel0(size_t n, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipDoubleComplex(0, 0);
    }
}

template <typename T>
void min_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, min_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    min_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void min_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, min_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    min_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void min_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, min_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    min_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

    hipDeviceSynchronize();
}

void egblas_smin(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        min_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        min_kernel0_run(n, y, incy);
    } else {
        min_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dmin(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        min_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        min_kernel0_run(n, y, incy);
    } else {
        min_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_cmin(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        min_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        min_kernel0_run(n, y, incy);
    } else {
        min_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zmin(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        min_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        min_kernel0_run(n, y, incy);
    } else {
        min_kernel_run(n, alpha, x, incx, y, incy);
    }
}
