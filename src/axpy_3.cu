#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/axpy_3.hpp"

#include "complex.hpp"

template <typename T>
__global__ void axpy_3_kernel(size_t n, T alpha, const T* x, size_t incx, const T* y, size_t incy, T* yy, size_t incyy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        yy[incyy * index] = alpha * x[incx * index] + y[incy * index];
    }
}

template <typename T>
__global__ void axpy_3_kernel1(size_t n, const T* x, size_t incx, const T* y, size_t incy, T* yy, size_t incyy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        yy[incyy * index] = x[incx * index] + y[incy * index];
    }
}

template <typename T>
__global__ void axpy_3_kernel0(size_t n, T* yy, size_t incyy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        yy[incyy * index] = zero<T>();
    }
}

template <typename T>
void axpy_3_kernel_run(size_t n, T alpha, const T* x, size_t incx, const T* y, size_t incy, T* yy, size_t incyy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, axpy_3_kernel<T>, 0, 0);

    int gridSize = ((n / incyy) + blockSize - 1) / blockSize;

    axpy_3_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy, yy, incyy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void axpy_3_kernel1_run(size_t n, const T* x, size_t incx, const T* y, size_t incy, T* yy, size_t incyy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, axpy_3_kernel1<T>, 0, 0);

    int gridSize = ((n / incyy) + blockSize - 1) / blockSize;

    axpy_3_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy, yy, incyy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void axpy_3_kernel0_run(size_t n, T* yy, size_t incyy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, axpy_3_kernel0<T>, 0, 0);

    int gridSize = ((n / incyy) + blockSize - 1) / blockSize;

    axpy_3_kernel0<T><<<gridSize, blockSize>>>(n, yy, incyy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_saxpy_3(size_t n, float alpha, const float* x, size_t incx, const float* y, size_t incy, float* yy, size_t incyy) {
    if (alpha == 1.0f) {
        axpy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha == 0.0f) {
        axpy_3_kernel0_run(n, yy, incyy);
    } else {
        axpy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

void egblas_daxpy_3(size_t n, double alpha, const double* x, size_t incx, const double* y, size_t incy, double* yy, size_t incyy) {
    if (alpha == 1.0) {
        axpy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha == 0.0) {
        axpy_3_kernel0_run(n, yy, incyy);
    } else {
        axpy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

void egblas_caxpy_3(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, const hipComplex* y, size_t incy, hipComplex* yy, size_t incyy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        axpy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        axpy_3_kernel0_run(n, yy, incyy);
    } else {
        axpy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}

void egblas_zaxpy_3(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, const hipDoubleComplex* y, size_t incy, hipDoubleComplex* yy, size_t incyy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        axpy_3_kernel1_run(n, x, incx, y, incy, yy, incyy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        axpy_3_kernel0_run(n, yy, incyy);
    } else {
        axpy_3_kernel_run(n, alpha, x, incx, y, incy, yy, incyy);
    }
}
