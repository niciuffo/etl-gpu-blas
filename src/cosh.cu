#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/cosh.hpp"

template <typename T>
__global__ void cosh_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * cosh(x[incx * index]);
    }
}

template <>
__global__ void cosh_kernel(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        auto res = make_hipComplex(cosh(c.x) * cos(c.y), sinh(c.x) * sin(c.y));

        y[incx * index] = hipCmulf(alpha, res);
    }
}

template <>
__global__ void cosh_kernel(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        auto res = make_hipDoubleComplex(cosh(c.x) * cos(c.y), sinh(c.x) * sin(c.y));

        y[incx * index] = hipCmul(alpha, res);
    }
}

template <typename T>
__global__ void cosh_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = cosh(x[incx * index]);
    }
}

template <>
__global__ void cosh_kernel1(size_t n, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        auto res = make_hipComplex(cosh(c.x) * cos(c.y), sinh(c.x) * sin(c.y));

        y[incx * index] = res;
    }
}

template <>
__global__ void cosh_kernel1(size_t n, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        auto res = make_hipDoubleComplex(cosh(c.x) * cos(c.y), sinh(c.x) * sin(c.y));

        y[incx * index] = res;
    }
}

template <typename T>
__global__ void cosh_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = T(0);
    }
}

template <>
__global__ void cosh_kernel0(size_t n, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipComplex(0, 0);
    }
}

template <>
__global__ void cosh_kernel0(size_t n, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipDoubleComplex(0, 0);
    }
}

template <typename T>
void cosh_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cosh_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    cosh_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void cosh_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cosh_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    cosh_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void cosh_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cosh_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    cosh_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_scosh(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        cosh_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        cosh_kernel0_run(n, y, incy);
    } else {
        cosh_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dcosh(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        cosh_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        cosh_kernel0_run(n, y, incy);
    } else {
        cosh_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_ccosh(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        cosh_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        cosh_kernel0_run(n, y, incy);
    } else {
        cosh_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zcosh(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        cosh_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        cosh_kernel0_run(n, y, incy);
    } else {
        cosh_kernel_run(n, alpha, x, incx, y, incy);
    }
}
