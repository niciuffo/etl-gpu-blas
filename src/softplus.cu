#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/softplus.hpp"
#include "complex.hpp"

template <typename T>
__forceinline__ __device__ T softplus(T x) {
    return log(T(1) + exp(x));
}

template <>
__forceinline__ __device__ hipComplex softplus(hipComplex x) {
    return log(hipCaddf(make_hipComplex(1,0), exp(x)));
}

template <>
__forceinline__ __device__ hipDoubleComplex softplus(hipDoubleComplex x) {
    return log(hipCadd(make_hipDoubleComplex(1,0), exp(x)));
}

template <typename T>
__global__ void softplus_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * softplus(x[incx * index]);
    }
}

template <typename T>
__global__ void softplus_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = softplus(x[incx * index]);
    }
}

template <typename T>
__global__ void softplus_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = T(0);
    }
}

template <>
__global__ void softplus_kernel0(size_t n, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipComplex(0, 0);
    }
}

template <>
__global__ void softplus_kernel0(size_t n, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipDoubleComplex(0, 0);
    }
}

template <typename T>
void softplus_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, softplus_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    softplus_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void softplus_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, softplus_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    softplus_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void softplus_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, softplus_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    softplus_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_ssoftplus(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        softplus_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        softplus_kernel0_run(n, y, incy);
    } else {
        softplus_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dsoftplus(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        softplus_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        softplus_kernel0_run(n, y, incy);
    } else {
        softplus_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_csoftplus(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        softplus_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        softplus_kernel0_run(n, y, incy);
    } else {
        softplus_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zsoftplus(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        softplus_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        softplus_kernel0_run(n, y, incy);
    } else {
        softplus_kernel_run(n, alpha, x, incx, y, incy);
    }
}
