#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

#include "egblas/assert.hpp"
#include "egblas/utils.hpp"
#include "egblas/batch_k_scale.hpp"
#include "egblas/cuda_check.hpp"

// 2D Version

template <typename T>
__global__ void batch_k_scale_plus2_kernel(size_t B, size_t K, const T* x, const T* gamma, const T* beta, T* y) {
    auto bk  = threadIdx.x + blockIdx.x * blockDim.x;

    if (bk < B * K) {
        const size_t b = bk / K;
        const size_t k = bk % K;

        y[b * K + k] = gamma[k] * x[b * K + k] + beta[k];
    }
}

template <typename T>
void egblas_batch_k_scale_plus2_run(size_t b, size_t k, const T* x, const T* gamma, const T* beta, T* y) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, batch_k_scale_plus2_kernel<T>, 0, 0);

    int gridSize = ((b * k) + blockSize - 1) / blockSize;

    batch_k_scale_plus2_kernel<<<gridSize, blockSize>>>(b, k, x, gamma, beta, y);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_sbatch_k_scale_plus2(size_t b, size_t k, const float* x, const float* gamma, const float* beta, float* y) {
    egblas_batch_k_scale_plus2_run(b, k, x, gamma, beta, y);
}

void egblas_dbatch_k_scale_plus2(size_t b, size_t k, const double* x, const double* gamma, const double* beta, double* y) {
    egblas_batch_k_scale_plus2_run(b, k, x, gamma, beta, y);
}

// 4D version

template <typename T>
__global__ void batch_k_scale_plus4_kernel(size_t B, size_t K, size_t M, size_t N, const T* x, const T* gamma, const T* beta, T* y) {
    auto bkmn  = threadIdx.x + blockIdx.x * blockDim.x;

    if (bkmn < B * K * M * N) {
        const size_t k = (bkmn / (M * N)) % K;

        y[bkmn] = gamma[k] * x[bkmn] + beta[k];
    }
}

template <typename T>
void egblas_batch_k_scale_plus4_run(size_t b, size_t k, size_t m, size_t n, const T* x, const T* gamma, const T* beta, T* y) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, batch_k_scale_plus4_kernel<T>, 0, 0);

    int gridSize = ((b * k * m * n) + blockSize - 1) / blockSize;

    batch_k_scale_plus4_kernel<<<gridSize, blockSize>>>(b, k, m, n, x, gamma, beta, y);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_sbatch_k_scale_plus4(size_t b, size_t k, size_t m, size_t n, const float* x, const float* gamma, const float* beta, float* y) {
    egblas_batch_k_scale_plus4_run(b, k, m, n, x, gamma, beta, y);
}

void egblas_dbatch_k_scale_plus4(size_t b, size_t k, size_t m, size_t n, const double* x, const double * gamma, const double* beta, double* y) {
    egblas_batch_k_scale_plus4_run(b, k, m, n, x, gamma, beta, y);
}
