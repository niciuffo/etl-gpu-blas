#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/ceil.hpp"

#include "complex.hpp"

template <typename T>
__global__ void ceil_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * ceil(x[incx * index]);
    }
}

template <typename T>
__global__ void ceil_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = ceil(x[incx * index]);
    }
}

template <typename T>
__global__ void ceil_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = zero<T>();
    }
}

template <typename T>
void ceil_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ceil_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    ceil_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void ceil_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ceil_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    ceil_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void ceil_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ceil_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    ceil_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_sceil(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        ceil_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        ceil_kernel0_run(n, y, incy);
    } else {
        ceil_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dceil(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        ceil_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        ceil_kernel0_run(n, y, incy);
    } else {
        ceil_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_cceil(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        ceil_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        ceil_kernel0_run(n, y, incy);
    } else {
        ceil_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zceil(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        ceil_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        ceil_kernel0_run(n, y, incy);
    } else {
        ceil_kernel_run(n, alpha, x, incx, y, incy);
    }
}
