#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

#include "egblas/assert.hpp"
#include "egblas/utils.hpp"
#include "egblas/sum.hpp"
#include "egblas/cuda_check.hpp"

#include "sum_reduce.hpp"

template <typename T>
__global__ void bias_batch_sum_kernel(size_t B, size_t N, const T* x, size_t incx, T* y, size_t incy) {
    auto n  = threadIdx.x + blockIdx.x * blockDim.x;

    if (n < N) {
        T sum = 0;

        for (size_t b = 0; b < B; ++b) {
            sum += x[(b * N + n) * incx];
        }

        y[incy * n] = sum;
    }
}

template <typename T>
__global__ void bias_batch_sum_kernel_flat(size_t B, size_t N, const T* x, T* y) {
    auto n  = threadIdx.x + blockIdx.x * blockDim.x;

    if (n < N) {
        T sum = 0;

        for (size_t b = 0; b < B; ++b) {
            sum += x[b * N + n];
        }

        y[n] = sum;
    }
}

void egblas_sbias_batch_sum(size_t b, size_t n, float* x, size_t incx, float* y, size_t incy) {
    const int blockSize = 64;
    const int gridSize = (n + blockSize - 1) / blockSize;

    if (incx == 1 && incy == 1) {
        bias_batch_sum_kernel_flat<<<gridSize, blockSize>>>(b, n, x, y);
    } else {
        bias_batch_sum_kernel<<<gridSize, blockSize>>>(b, n, x, incx, y, incy);
    }
}

void egblas_dbias_batch_sum(size_t b, size_t n, double* x, size_t incx, double* y, size_t incy) {
    const int blockSize = 64;
    const int gridSize = (n + blockSize - 1) / blockSize;

    if (incx == 1 && incy == 1) {
        bias_batch_sum_kernel_flat<<<gridSize, blockSize>>>(b, n, x, y);
    } else {
        bias_batch_sum_kernel<<<gridSize, blockSize>>>(b, n, x, incx, y, incy);
    }
}
