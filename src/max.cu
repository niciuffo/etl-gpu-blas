#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/max.hpp"

template <typename T>
__global__ void max_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * max(x[incx * index], y[incy * index]);
    }
}

template <>
__global__ void max_kernel(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto x_ =  x[incx * index];
        auto y_ =  y[incy * index];

        if (x_.x > y_.x) {
            y[incy * index] = hipCmulf(alpha, x_);
        } else if (y_.x > x_.x) {
            y[incy * index] = hipCmulf(alpha, y_);
        } else {
            if (x_.y > y_.y) {
                y[incy * index] = hipCmulf(alpha, x_);
            } else {
                y[incy * index] = hipCmulf(alpha, y_);
            }
        }
    }
}

template <>
__global__ void max_kernel(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto x_ =  x[incx * index];
        auto y_ =  y[incy * index];

        if (x_.x > y_.x) {
            y[incy * index] = hipCmul(alpha, x_);
        } else if (y_.x > x_.x) {
            y[incy * index] = hipCmul(alpha, y_);
        } else {
            if (x_.y > y_.y) {
                y[incy * index] = hipCmul(alpha, x_);
            } else {
                y[incy * index] = hipCmul(alpha, y_);
            }
        }
    }
}

template <typename T>
__global__ void max_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = max(x[incx * index], y[incy * index]);
    }
}

template <>
__global__ void max_kernel1(size_t n, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto x_ =  x[incx * index];
        auto y_ =  y[incy * index];

        if (x_.x > y_.x) {
            y[incy * index] = x_;
        } else if (y_.x > x_.x) {
            y[incy * index] = y_;
        } else {
            if (x_.y > y_.y) {
                y[incy * index] = x_;
            } else {
                y[incy * index] = y_;
            }
        }
    }
}

template <>
__global__ void max_kernel1(size_t n, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto x_ =  x[incx * index];
        auto y_ =  y[incy * index];

        if (x_.x > y_.x) {
            y[incy * index] = x_;
        } else if (y_.x > x_.x) {
            y[incy * index] = y_;
        } else {
            if (x_.y > y_.y) {
                y[incy * index] = x_;
            } else {
                y[incy * index] = y_;
            }
        }
    }
}

template <typename T>
__global__ void max_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = T(0);
    }
}

template <>
__global__ void max_kernel0(size_t n, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipComplex(0, 0);
    }
}

template <>
__global__ void max_kernel0(size_t n, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipDoubleComplex(0, 0);
    }
}

template <typename T>
void max_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, max_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    max_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void max_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, max_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    max_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void max_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, max_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    max_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

    hipDeviceSynchronize();
}

void egblas_smax(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        max_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        max_kernel0_run(n, y, incy);
    } else {
        max_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dmax(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        max_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        max_kernel0_run(n, y, incy);
    } else {
        max_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_cmax(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        max_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        max_kernel0_run(n, y, incy);
    } else {
        max_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zmax(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        max_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        max_kernel0_run(n, y, incy);
    } else {
        max_kernel_run(n, alpha, x, incx, y, incy);
    }
}
