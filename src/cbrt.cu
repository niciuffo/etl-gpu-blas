#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/cbrt.hpp"

#include "complex.hpp"

template <typename T>
__global__ void cbrt_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * cbrt(x[incx * index]);
    }
}

template <>
__global__ void cbrt_kernel(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto c = x[incx * index];

        auto res = cbrt(c);

        y[incy * index] = hipCmulf(alpha, res);
    }
}

template <>
__global__ void cbrt_kernel(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto c = x[incx * index];

        auto res = cbrt(c);

        y[incy * index] = hipCmul(alpha, res);
    }
}

template <typename T>
__global__ void cbrt_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = cbrt(x[incx * index]);
    }
}

template <>
__global__ void cbrt_kernel1(size_t n, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto c = x[incx * index];

        auto res = cbrt(c);

        y[incy * index] = res;
    }
}

template <>
__global__ void cbrt_kernel1(size_t n, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        auto c = x[incx * index];

        auto res = cbrt(c);

        y[incy * index] = res;
    }
}

template <typename T>
__global__ void cbrt_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = T(0);
    }
}

template <>
__global__ void cbrt_kernel0(size_t n, hipComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipComplex(0, 0);
    }
}

template <>
__global__ void cbrt_kernel0(size_t n, hipDoubleComplex* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = make_hipDoubleComplex(0, 0);
    }
}

template <typename T>
void cbrt_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cbrt_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    cbrt_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void cbrt_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cbrt_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    cbrt_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

template <typename T>
void cbrt_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cbrt_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    cbrt_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_scbrt(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        cbrt_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        cbrt_kernel0_run(n, y, incy);
    } else {
        cbrt_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dcbrt(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        cbrt_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        cbrt_kernel0_run(n, y, incy);
    } else {
        cbrt_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_ccbrt(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        cbrt_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        cbrt_kernel0_run(n, y, incy);
    } else {
        cbrt_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zcbrt(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        cbrt_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        cbrt_kernel0_run(n, y, incy);
    } else {
        cbrt_kernel_run(n, alpha, x, incx, y, incy);
    }
}
