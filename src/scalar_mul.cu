#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/scalar_mul.hpp"

template <typename T>
__global__ void scalar_mul_kernel(T* x, size_t n, size_t s, const T beta) {
    auto index  = 1 * (threadIdx.x + blockIdx.x * blockDim.x);
    auto stride = 1 * (blockDim.x * gridDim.x);

    for (; index < n; index += stride) {
        x[s * index] *= beta;
    }
}

template <>
__global__ void scalar_mul_kernel(hipDoubleComplex* x, size_t n, size_t s, const hipDoubleComplex beta) {
    auto index  = 1 * (threadIdx.x + blockIdx.x * blockDim.x);
    auto stride = 1 * (blockDim.x * gridDim.x);

    for (; index < n; index += stride) {
        x[s * index] = hipCmul(x[s * index], beta);
    }
}

template <>
__global__ void scalar_mul_kernel(hipComplex* x, size_t n, size_t s, const hipComplex beta) {
    auto index  = 1 * (threadIdx.x + blockIdx.x * blockDim.x);
    auto stride = 1 * (blockDim.x * gridDim.x);

    for (; index < n; index += stride) {
        x[s * index] = hipCmulf(x[s * index], beta);
    }
}

template <typename T>
void scalar_mul_kernel_run(T* x, size_t n, size_t s, T beta) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, scalar_mul_kernel<T>, 0, 0);

    int gridSize = ((n / s) + blockSize - 1) / blockSize;

    scalar_mul_kernel<T><<<gridSize, blockSize>>>(x, n, s, beta);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_scalar_smul(float* x, size_t n, size_t s, float beta) {
    scalar_mul_kernel_run(x, n, s, beta);
}

void egblas_scalar_dmul(double* x, size_t n, size_t s, double beta) {
    scalar_mul_kernel_run(x, n, s, beta);
}

void egblas_scalar_cmul(hipComplex* x, size_t n, size_t s, hipComplex beta) {
    scalar_mul_kernel_run(x, n, s, beta);
}

void egblas_scalar_zmul(hipDoubleComplex* x, size_t n, size_t s, hipDoubleComplex beta) {
    scalar_mul_kernel_run(x, n, s, beta);
}
