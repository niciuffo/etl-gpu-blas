#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/scalar_mul.hpp"

template <typename T>
__global__ void scalar_mul_kernel(T* x, size_t n, size_t s, const T beta) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        x[s * index] *= beta;
    }
}

template <typename T>
__global__ void scalar_mul_kernel_flat(T* x, size_t n, const T beta) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        x[index] *= beta;
    }
}

template <>
__global__ void scalar_mul_kernel(hipDoubleComplex* x, size_t n, size_t s, const hipDoubleComplex beta) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        x[s * index] = hipCmul(x[s * index], beta);
    }
}

template <>
__global__ void scalar_mul_kernel_flat(hipDoubleComplex* x, size_t n, const hipDoubleComplex beta) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        x[index] = hipCmul(x[index], beta);
    }
}

template <>
__global__ void scalar_mul_kernel(hipComplex* x, size_t n, size_t s, const hipComplex beta) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        x[s * index] = hipCmulf(x[s * index], beta);
    }
}

template <>
__global__ void scalar_mul_kernel_flat(hipComplex* x, size_t n, const hipComplex beta) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        x[index] = hipCmulf(x[index], beta);
    }
}

template <typename T>
void scalar_mul_kernel_run(T* x, size_t n, size_t s, T beta) {
    static int blockSize   = 0;
    static int minGridSize = 0;

    if (!blockSize) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, scalar_mul_kernel<T>, 0, 0);
    }

    int gridSize = (n + blockSize - 1) / blockSize;

    if (s == 1) {
        scalar_mul_kernel_flat<T><<<gridSize, blockSize>>>(x, n, beta);
    } else {
        scalar_mul_kernel<T><<<gridSize, blockSize>>>(x, n, s, beta);
    }

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_scalar_smul(float* x, size_t n, size_t s, float beta) {
    scalar_mul_kernel_run(x, n, s, beta);
}

void egblas_scalar_dmul(double* x, size_t n, size_t s, double beta) {
    scalar_mul_kernel_run(x, n, s, beta);
}

void egblas_scalar_cmul(hipComplex* x, size_t n, size_t s, hipComplex beta) {
    scalar_mul_kernel_run(x, n, s, beta);
}

void egblas_scalar_zmul(hipDoubleComplex* x, size_t n, size_t s, hipDoubleComplex beta) {
    scalar_mul_kernel_run(x, n, s, beta);
}
