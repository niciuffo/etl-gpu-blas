#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/real.hpp"

template <typename T, typename TT>
__global__ void real_kernel(size_t n, TT alpha, const T* x, size_t incx, TT* y, size_t incy);

template <>
__global__ void real_kernel(size_t n, float alpha, const hipComplex* x, size_t incx, float* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        y[incx * index] = alpha * c.x;
    }
}

template <>
__global__ void real_kernel(size_t n, double alpha, const hipDoubleComplex* x, size_t incx, double* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        y[incx * index] = alpha * c.x;
    }
}

template <typename T, typename TT>
__global__ void real_kernel1(size_t n, const T* x, size_t incx, TT* y, size_t incy);

template <>
__global__ void real_kernel1(size_t n, const hipComplex* x, size_t incx, float* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        y[incx * index] = c.x;
    }
}

template <>
__global__ void real_kernel1(size_t n, const hipDoubleComplex* x, size_t incx, double* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        y[incx * index] = c.x;
    }
}

template <typename T>
__global__ void real_kernel0(size_t n, T* y, size_t incy){
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = 0;
    }
}

template <typename T, typename TT>
void real_kernel_run(size_t n, TT alpha, const T* x, size_t incx, TT* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, real_kernel<T, TT>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    real_kernel<T, TT><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T, typename TT>
void real_kernel1_run(size_t n, const T* x, size_t incx, TT* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, real_kernel1<T, TT>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    real_kernel1<T, TT><<<gridSize, blockSize>>>(n, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void real_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, real_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    real_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

    hipDeviceSynchronize();
}

void egblas_creal(size_t n, float alpha, const hipComplex* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        real_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        real_kernel0_run(n, y, incy);
    } else {
        real_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zreal(size_t n, double alpha, const hipDoubleComplex* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        real_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        real_kernel0_run(n, y, incy);
    } else {
        real_kernel_run(n, alpha, x, incx, y, incy);
    }
}
