#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/not_equal.hpp"

#include "complex.hpp"

__device__ bool operator!=(hipComplex lhs, hipComplex rhs){
    return lhs.x != rhs.x || lhs.y != rhs.y;
}

__device__ bool operator!=(hipDoubleComplex lhs, hipDoubleComplex rhs){
    return lhs.x != rhs.x || lhs.y != rhs.y;
}

template <typename T>
__global__ void not_equal_kernel(size_t n, const T* a, size_t inca, const T* b, size_t incb, bool* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = a[inca * index] != b[incb * index];
    }
}

template <typename T>
void not_equal_kernel_run(size_t n, const T* a, size_t inca, const T* b, size_t incb, bool* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, not_equal_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    not_equal_kernel<T><<<gridSize, blockSize>>>(n, a, inca, b, incb, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_snot_equal(size_t n, const float* a, size_t inca, const float* b, size_t incb, bool* y, size_t incy) {
    not_equal_kernel_run(n, a, inca, b, incb, y, incy);
}

void egblas_dnot_equal(size_t n, const double* a, size_t inca, const double* b, size_t incb, bool* y, size_t incy) {
    not_equal_kernel_run(n, a, inca, b, incb, y, incy);
}

void egblas_cnot_equal(size_t n, const hipComplex* a, size_t inca, const hipComplex* b, size_t incb, bool* y, size_t incy) {
    not_equal_kernel_run(n, a, inca, b, incb, y, incy);
}

void egblas_znot_equal(size_t n, const hipDoubleComplex* a, size_t inca, const hipDoubleComplex* b, size_t incb, bool* y, size_t incy) {
    not_equal_kernel_run(n, a, inca, b, incb, y, incy);
}
