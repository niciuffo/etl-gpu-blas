#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/scalar_add.hpp"

template <typename T>
__global__ void scalar_add_kernel(T* x, size_t n, size_t s, const T beta) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        x[s * index] += beta;
    }
}

template <typename T>
__global__ void scalar_add_kernel_flat(T* x, size_t n, const T beta) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        x[index] += beta;
    }
}

template <>
__global__ void scalar_add_kernel(hipDoubleComplex* x, size_t n, size_t s, const hipDoubleComplex beta) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        x[s * index] = hipCadd(x[s * index], beta);
    }
}

template <>
__global__ void scalar_add_kernel_flat(hipDoubleComplex* x, size_t n, const hipDoubleComplex beta) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        x[index] = hipCadd(x[index], beta);
    }
}

template <>
__global__ void scalar_add_kernel(hipComplex* x, size_t n, size_t s, const hipComplex beta) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        x[s * index] = hipCaddf(x[s * index], beta);
    }
}

template <>
__global__ void scalar_add_kernel_flat(hipComplex* x, size_t n, const hipComplex beta) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        x[index] = hipCaddf(x[index], beta);
    }
}

template <typename T>
void scalar_add_kernel_run(T* x, size_t n, size_t s, T beta) {
    static int blockSize   = 0;
    static int minGridSize = 0;

    if (!blockSize) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, scalar_add_kernel<T>, 0, 0);
    }

    int gridSize = (n + blockSize - 1) / blockSize;

    if (s == 1) {
        scalar_add_kernel_flat<T><<<gridSize, blockSize>>>(x, n, beta);
    } else {
        scalar_add_kernel<T><<<gridSize, blockSize>>>(x, n, s, beta);
    }

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_scalar_sadd(float* x, size_t n, size_t s, float beta) {
    scalar_add_kernel_run(x, n, s, beta);
}

void egblas_scalar_dadd(double* x, size_t n, size_t s, double beta) {
    scalar_add_kernel_run(x, n, s, beta);
}

void egblas_scalar_cadd(hipComplex* x, size_t n, size_t s, hipComplex beta) {
    scalar_add_kernel_run(x, n, s, beta);
}

void egblas_scalar_zadd(hipDoubleComplex* x, size_t n, size_t s, hipDoubleComplex beta) {
    scalar_add_kernel_run(x, n, s, beta);
}
