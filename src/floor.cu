#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/floor.hpp"

#include "complex.hpp"

template <typename T>
__global__ void floor_kernel(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = alpha * floor(x[incx * index]);
    }
}

template <typename T>
__global__ void floor_kernel1(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = floor(x[incx * index]);
    }
}

template <typename T>
__global__ void floor_kernel0(size_t n, T* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = zero<T>();
    }
}

template <typename T>
void floor_kernel_run(size_t n, T alpha, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, floor_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    floor_kernel<T><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void floor_kernel1_run(size_t n, const T* x, size_t incx, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, floor_kernel1<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    floor_kernel1<T><<<gridSize, blockSize>>>(n, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void floor_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, floor_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    floor_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

    hipDeviceSynchronize();
}

void egblas_sfloor(size_t n, float alpha, const float* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        floor_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        floor_kernel0_run(n, y, incy);
    } else {
        floor_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_dfloor(size_t n, double alpha, const double* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        floor_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        floor_kernel0_run(n, y, incy);
    } else {
        floor_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_cfloor(size_t n, hipComplex alpha, const hipComplex* x, size_t incx, hipComplex* y, size_t incy) {
    if (alpha.x == 1.0f && alpha.y == 0.0f) {
        floor_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0f && alpha.y == 0.0f) {
        floor_kernel0_run(n, y, incy);
    } else {
        floor_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zfloor(size_t n, hipDoubleComplex alpha, const hipDoubleComplex* x, size_t incx, hipDoubleComplex* y, size_t incy) {
    if (alpha.x == 1.0 && alpha.y == 0.0) {
        floor_kernel1_run(n, x, incx, y, incy);
    } else if (alpha.x == 0.0 && alpha.y == 0.0) {
        floor_kernel0_run(n, y, incy);
    } else {
        floor_kernel_run(n, alpha, x, incx, y, incy);
    }
}
