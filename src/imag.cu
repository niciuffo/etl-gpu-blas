#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/imag.hpp"

template <typename T, typename TT>
__global__ void imag_kernel(size_t n, TT alpha, const T* x, size_t incx, TT* y, size_t incy);

template <>
__global__ void imag_kernel(size_t n, float alpha, const hipComplex* x, size_t incx, float* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        y[incx * index] = alpha * c.y;
    }
}

template <>
__global__ void imag_kernel(size_t n, double alpha, const hipDoubleComplex* x, size_t incx, double* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        y[incx * index] = alpha * c.y;
    }
}

template <typename T, typename TT>
__global__ void imag_kernel1(size_t n, const T* x, size_t incx, TT* y, size_t incy);

template <>
__global__ void imag_kernel1(size_t n, const hipComplex* x, size_t incx, float* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipComplex c = x[incx * index];

        y[incx * index] = c.y;
    }
}

template <>
__global__ void imag_kernel1(size_t n, const hipDoubleComplex* x, size_t incx, double* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        hipDoubleComplex c = x[incx * index];

        y[incx * index] = c.y;
    }
}

template <typename T>
__global__ void imag_kernel0(size_t n, T* y, size_t incy){
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = 0;
    }
}

template <typename T, typename TT>
void imag_kernel_run(size_t n, TT alpha, const T* x, size_t incx, TT* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, imag_kernel<T, TT>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    imag_kernel<T, TT><<<gridSize, blockSize>>>(n, alpha, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T, typename TT>
void imag_kernel1_run(size_t n, const T* x, size_t incx, TT* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, imag_kernel1<T, TT>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    imag_kernel1<T, TT><<<gridSize, blockSize>>>(n, x, incx, y, incy);

    hipDeviceSynchronize();
}

template <typename T>
void imag_kernel0_run(size_t n, T* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, imag_kernel0<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    imag_kernel0<T><<<gridSize, blockSize>>>(n, y, incy);

    hipDeviceSynchronize();
}

void egblas_cimag(size_t n, float alpha, const hipComplex* x, size_t incx, float* y, size_t incy) {
    if (alpha == 1.0f) {
        imag_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0f) {
        imag_kernel0_run(n, y, incy);
    } else {
        imag_kernel_run(n, alpha, x, incx, y, incy);
    }
}

void egblas_zimag(size_t n, double alpha, const hipDoubleComplex* x, size_t incx, double* y, size_t incy) {
    if (alpha == 1.0) {
        imag_kernel1_run(n, x, incx, y, incy);
    } else if (alpha == 0.0) {
        imag_kernel0_run(n, y, incy);
    } else {
        imag_kernel_run(n, alpha, x, incx, y, incy);
    }
}
