#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/less_equal.hpp"

#include "complex.hpp"

__device__ bool operator<=(hipComplex lhs, hipComplex rhs){
    return lhs.x < rhs.x || (lhs.x == rhs.x && lhs.y <= rhs.y);
}

__device__ bool operator<=(hipDoubleComplex lhs, hipDoubleComplex rhs){
    return lhs.x < rhs.x || (lhs.x == rhs.x && lhs.y <= rhs.y);
}

template <typename T>
__global__ void less_equal_kernel(size_t n, const T* a, size_t inca, const T* b, size_t incb, bool* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = a[inca * index] <= b[incb * index];
    }
}

template <typename T>
void less_equal_kernel_run(size_t n, const T* a, size_t inca, const T* b, size_t incb, bool* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, less_equal_kernel<T>, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    less_equal_kernel<T><<<gridSize, blockSize>>>(n, a, inca, b, incb, y, incy);

    hipDeviceSynchronize();
}

void egblas_sless_equal(size_t n, const float* a, size_t inca, const float* b, size_t incb, bool* y, size_t incy) {
    less_equal_kernel_run(n, a, inca, b, incb, y, incy);
}

void egblas_dless_equal(size_t n, const double* a, size_t inca, const double* b, size_t incb, bool* y, size_t incy) {
    less_equal_kernel_run(n, a, inca, b, incb, y, incy);
}

void egblas_cless_equal(size_t n, const hipComplex* a, size_t inca, const hipComplex* b, size_t incb, bool* y, size_t incy) {
    less_equal_kernel_run(n, a, inca, b, incb, y, incy);
}

void egblas_zless_equal(size_t n, const hipDoubleComplex* a, size_t inca, const hipDoubleComplex* b, size_t incb, bool* y, size_t incy) {
    less_equal_kernel_run(n, a, inca, b, incb, y, incy);
}
