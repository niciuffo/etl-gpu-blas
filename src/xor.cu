#include "hip/hip_runtime.h"
//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include "egblas/xor.hpp"

__global__ void xor_kernel(size_t n, const bool* a, size_t inca, const bool* b, size_t incb, bool* y, size_t incy) {
    auto index  = threadIdx.x + blockIdx.x * blockDim.x;
    auto stride = blockDim.x * gridDim.x;

    for (; index < n; index += stride) {
        y[incy * index] = a[inca * index] != b[incb * index];
    }
}

void xor_kernel_run(size_t n, const bool* a, size_t inca, const bool* b, size_t incb, bool* y, size_t incy) {
    int blockSize;
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, xor_kernel, 0, 0);

    int gridSize = ((n / incy) + blockSize - 1) / blockSize;

    xor_kernel<<<gridSize, blockSize>>>(n, a, inca, b, incb, y, incy);

#ifdef EGBLAS_SYNCHRONIZE
    hipDeviceSynchronize();
#endif
}

void egblas_bxor(size_t n, const bool* a, size_t inca, const bool* b, size_t incb, bool* y, size_t incy) {
    xor_kernel_run(n, a, inca, b, incb, y, incy);
}
